#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void cell(int *univ, int h, int w)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int num_cells = h * w;
	// declare univ

	for (id; id < num_cells; id += blockDim.x * gridDim.x) 
	{
		int x_pos = id % w;
		int y_pos = id - x_pos;
		int x_left = (x + w - 1) % w;
		int x_right = (x + 1) % w;
		int y_above = (y_pos + num_cells - w) % num_cells;
		int y_below = (y_pos + num_cells) % num_cells;

		int alive = univ[y_above][x_left] + univ[y_above][x_pos] + univ[y_above][x_right] +
								univ[y_pos][x_left] + univ[y_pos][x_right] +
								univ[y_below][x_left] + univ[y_below][x_pos] + univ[y_below][x_right];

		new_univ[y_pos][x_pos] = alive == 3 || (alive == 2 && univ[y_pos][x_pos]) ? 1 : 0;
	}
}

void print_matrix(int *univ, int h, int w)
{
	// http://www.geeksforgeeks.org/pass-2d-array-parameter-c/
	// Credit to the above for demonstrating how to pass around 2d arrays
	
  printf("\n");

  for (int y = 0; y < h; y++)
  {
    for (int x = 0; x < w; x++)
    {
      printf("%d", *((univ + y * w) + x));
    }

    printf("\n");
  }
}

void generate(int g, int h, int w)
{
	int univ[h][w];
	
	for (int y = 0; y < h; y++) 
	{
		for (int x = 0; x < w; x++) 
		{
			univ[y][x] = rand() % 2;
		}
	}

	while(g > 0)
	{
		// TODO		

		print_matrix((int *)univ, h, w);
		g--;
	}
}

int main()
{
	int g, h, w;

	printf("Enter desired number of generations:\n");
	scanf("%d", &g);

	printf("Enter desired height of universe:\n");
	scanf("%d", &h);

	printf("Enter desired width of universe:\n");
	scanf("%d", &w);

	generate(g, h, w);
}
