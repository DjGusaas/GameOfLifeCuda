#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define NUM_THREADS 256

#define min(a, b) (((a) < (b)) ? (a) : (b))
__global__ void cell_kernel(int univ[], int h, int w, int new_univ[])
{
	int id = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int size = h * w;

	for (id; id < size; id += blockDim.x * gridDim.x) 
	{
		// Neighbor positions
		unsigned x_pos = id % w;
		unsigned y_pos = id - x_pos;
		unsigned x_left = (x_pos + w - 1) % w;
		unsigned x_right = (x_pos + 1) % w;
		unsigned y_above = (y_pos + size - w) % size;
		unsigned y_below = (y_pos + size) % size;
		
		// Calculate number of alive neighbors
		int alive = univ[x_left + y_above] + univ[x_pos + y_above] + univ[x_right + y_above] + univ[x_left + y_pos] + univ[x_right + y_pos] + univ[x_left + y_below] + univ[x_pos + y_below] + univ[x_right + y_below];
		new_univ[x_pos + y_pos] = alive == 3 || (alive == 2 && univ[x_pos + y_pos]) ? 1 : 0;
	}
}

void print_array(int arr[], int size)
{	
  printf("\n");

  for (int i = 0; i < size; i++)
  {
    printf("%d", arr[i]);
  }

  printf("\n");
}
void generate(int g, int h, int w)
{
	// Number of cells in universe
	int size = h * w;

	// Host(CPU) arrays
	int h_univ[size];
	int h_new_univ[size];

	// Devide(GPU) arrays
	int d_univ[size];
	int d_new_univ[size]

	// "Randomly" seed universe
	for (int i = 0; i < size; i++) 
	{
		h_univ[i] = rand() % 2;
	}

	while(g > 0)
	{
		// Rename t to something more descriptive
		size_t t = (size) / NUM_THREADS;
		unsigned blocks_count = (unsigned)min((size_t)32768, t);

		cell_kernel<<<blocks_count, t>>>(d_univ, h, w, d_new_univ);

		// Perform some sort memory copying from GPU to CPU

		//memcpy(h_new_univ, h_univ, size);
		print_array(h_univ, size);
		g--;
	}

	// Release memory? 
}

int main()
{
	int g, h, w;

	printf("Enter desired number of generations:\n");
	scanf("%d", &g);

	printf("Enter desired height of universe:\n");
	scanf("%d", &h);

	printf("Enter desired width of universe:\n");
	scanf("%d", &w);

	generate(g, h, w);

	return 0;
}
