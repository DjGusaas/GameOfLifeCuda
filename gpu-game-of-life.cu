#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void cell_kernel(int univ[], int h, int w)
{
	int id = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int num_cells = h * w;

	for (id; id < num_cells; id += blockDim.x * gridDim.x) 
	{
		unsigned x_pos = id % w;
		unsigned y_pos = id - x_pos;
		unsigned x_left = (x_pos + w - 1) % w;
		unsigned x_right = (x_pos + 1) % w;
		unsigned y_above = (y_pos + num_cells - w) % num_cells;
		unsigned y_below = (y_pos + num_cells) % num_cells;

		int alive = univ[x_left + y_above] + univ[x_pos + y_above] + univ[x_right + y_above] + univ[x_left + y_pos] + univ[x_right + y_pos] + univ[x_left + y_below] + univ[x_pos + y_below] + univ[x_right + y_below];

		new_univ[x_pos + y_pos] = alive == 3 || (alive == 2 && univ[x_pos + y_pos]) ? 1 : 0;
	}
}

void print_array(int univ[], int size)
{	
  printf("\n");

  for (int i = 0; i < size; i++)
  {
    printf("%d", univ[i]);
  }

  printf("\n");
}

void generate(int g, int h, int w)
{
	int size = h * w;
	int univ[size];
	
	for (int i = 0; i < size; i++) 
	{
		univ[i] = rand() % 2;
	}

	while(g > 0)
	{
		size_t t = (size) / 256;
		unsigned blocks_count = (unsigned)std::min((size_t)32768, t);

		d_ ;
		d_ ;

		cell_kernel<<<blocks_count, 256>>>( , unsigned(w), unsigned(h), );

		print_array(univ, size);
		g--;
	}
}

int main()
{
	int g, h, w;

	printf("Enter desired number of generations:\n");
	scanf("%d", &g);

	printf("Enter desired height of universe:\n");
	scanf("%d", &h);

	printf("Enter desired width of universe:\n");
	scanf("%d", &w);

	generate(g, h, w);
}
