
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void cell()
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	// TODO
}

void print_matrix(int *univ, int h, int w)
{
	// http://www.geeksforgeeks.org/pass-2d-array-parameter-c/
	// Credit to the above for demonstrating how to pass around 2d arrays
	
  printf("\n");

  for (int y = 0; y < h; y++)
  {
    for (int x = 0; x < w; x++)
    {
      printf("%d", *((univ + y * w) + x));
    }

    printf("\n");
  }
}

void generate(int g, int h, int w)
{
	int univ[h][w];
	
	for (int y = 0; y < 3; y++) 
	{
		for (int x = 0; x < 3; x++) 
		{
			univ[y][x] = rand() % 2;
		}
	}

	while(g > 0)
	{
		// TODO
		print_matrix((int *)univ, h, w);
		g--;
	}
}

int main()
{
	int g, h, w;

	printf("Enter desired number of generations:\n");
	scanf("%d", &g);

	printf("Enter desired height of universe:\n");
	scanf("%d", &h);

	printf("Enter desired width of universe:\n");
	scanf("%d", &w);

	generate(g, h, w);
}
